#include "hip/hip_runtime.h"
﻿#include <engine/cuda/kernel.cuh>
#include <engine/board.h>
#include ""
#include <stdio.h>
#include <malloc.h>
#include <iostream>

__global__ void gpu::calculateKernel(bool* boxesStatusIn, bool* boxesStatusOut, unsigned long long int sizeOfArray)
{
	
}

void gpu::flipCellStatus(unsigned long long int x)
{
	hipMemcpy(cellsStatusBuffer_p, dev_cellsStatusIn_p, cellsStatusLength * sizeof(bool), hipMemcpyDeviceToHost);
	cellsStatusBuffer_p[x] = !cellsStatusBuffer_p[x];
	hipMemcpy(dev_cellsStatusIn_p, cellsStatusBuffer_p, cellsStatusLength * sizeof(bool), hipMemcpyHostToDevice);
}

void gpu::freeMemory()
{
	fprintf(stderr, "cuda free memory!");
	hipFree(dev_cellsStatusOut_p);
	hipFree(dev_cellsStatusIn_p);
	free(cellsStatusBuffer_p);

	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
}

hipError_t gpu::mallocMemory(unsigned long long int size)
{
	
	cellsStatusBuffer_p = (bool*)calloc(size, sizeof(bool));
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		freeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_cellsStatusIn_p, size * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_cellsStatusOut_p, size * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory();
		return cudaStatus;
	}

	bool* statusArray = (bool*)calloc(size, sizeof(bool));
	cudaStatus = hipMemcpy(dev_cellsStatusIn_p, statusArray, size * sizeof(bool), hipMemcpyHostToDevice);
	free(statusArray);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory();
		return cudaStatus;
	}

	return hipSuccess;
}

hipError_t gpu::calculateWithCuda(bool* statusArray)
{
	hipError_t cudaStatus;
	unsigned long long int size = cellsStatusLength;
	calculateKernel << <size, 8 >> > (dev_cellsStatusIn_p, dev_cellsStatusOut_p, size);


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freeMemory();
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(statusArray, dev_cellsStatusOut_p, size * sizeof(bool), hipMemcpyDeviceToHost);

	bool* buf = dev_cellsStatusIn_p;
	dev_cellsStatusIn_p = dev_cellsStatusOut_p;
	dev_cellsStatusOut_p = buf;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory();
		return cudaStatus;
	}

	return cudaStatus;
}
