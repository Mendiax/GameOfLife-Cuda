#include "hip/hip_runtime.h"
﻿#include <engine/cuda/kernel.cuh>
#include <engine/board.h>


//#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <malloc.h>
#include <iostream>

__global__ void gpu::calculateKernel(bool* boxesStatusIn, bool* boxesStatusOut, unsigned long long int sizeOfArray)
{
	
}


void gpu::flipCellStatus(unsigned long long int x)
{
	hipMemcpy(cellsStatusBuffer_p, dev_cellsStatusIn_p, cellsStatusLength * sizeof(bool), hipMemcpyDeviceToHost);
	cellsStatusBuffer_p[x] = !cellsStatusBuffer_p[x];
	hipMemcpy(dev_cellsStatusIn_p, cellsStatusBuffer_p, cellsStatusLength * sizeof(bool), hipMemcpyHostToDevice);
}

void gpu::freeMemory()
{
	fprintf(stderr, "cuda free memory!");
	hipFree(dev_cellsStatusOut_p);
	hipFree(dev_cellsStatusIn_p);
	free(cellsStatusBuffer_p);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
}

hipError_t gpu::mallocMemory(unsigned long long int size)
{
	
	cellsStatusBuffer_p = (bool*)calloc(size, sizeof(bool));
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		freeMemory();
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_cellsStatusIn_p, size * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_cellsStatusOut_p, size * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory();
		return cudaStatus;
	}

	bool* statusArray = (bool*)calloc(size, sizeof(bool));
	cudaStatus = hipMemcpy(dev_cellsStatusIn_p, statusArray, size * sizeof(bool), hipMemcpyHostToDevice);
	free(statusArray);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory();
		return cudaStatus;
	}

	return hipSuccess;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t gpu::calculateWithCuda(bool* statusArray)
{
	hipError_t cudaStatus;
	unsigned long long int size = cellsStatusLength;
	calculateKernel << <size, 8 >> > (dev_cellsStatusIn_p, dev_cellsStatusOut_p, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freeMemory();
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freeMemory();
		return cudaStatus;
	}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(statusArray, dev_cellsStatusOut_p, size * sizeof(bool), hipMemcpyDeviceToHost);

	bool* buf = dev_cellsStatusIn_p;
	dev_cellsStatusIn_p = dev_cellsStatusOut_p;
	dev_cellsStatusOut_p = buf;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory();
		return cudaStatus;
	}

	return cudaStatus;
}
