#include "hip/hip_runtime.h"
﻿#include <engine/cuda/kernel.cuh>
#include <engine/board.h>
#include ""
#include <stdio.h>
#include <malloc.h>
#include <iostream>

__device__ uint16_t rowCount;
__device__ uint16_t sizeOfBoxArray;


__device__ void moveUp(int* id)
{
	if (*id + rowCount < sizeOfBoxArray)
		*id += rowCount;
	else
		*id %= rowCount;
}
__device__ void moveDown(int* id)
{
	if (*id - rowCount >= 0)
		*id -= rowCount;
	else
		*id -= rowCount + sizeOfBoxArray;
}
__device__ void moveLeft(int* id)
{
	if (0 == *id % rowCount)
		*id = *id - 1 + rowCount;
	else
		*id -= 1;
}
__device__ void moveRight(int* id)
{
	if (rowCount - 1 == *id % rowCount)
		*id = *id + 1 - rowCount;
	else
		*id += 1;
}

__global__ void gpu::calculateKernel(bool* boxesStatusIn, bool* boxesStatusOut, unsigned long long int row, unsigned long long int sizeOfArray, bool* lifeArray, bool* deathArray)
{
	rowCount = row;
	sizeOfBoxArray = sizeOfArray;
	__shared__ int boxId;
	__shared__ unsigned int sum;
	__shared__ bool state;
	__shared__ int id[8];

	if (threadIdx.x == 0)
	{
		boxId = blockIdx.x;
		state = boxesStatusIn[boxId];
		sum = 0;
	}

	__syncthreads();

	id[threadIdx.x] = boxId;



	__shared__  bool surroundState[8];
	switch (threadIdx.x)
	{
	case 0:
		moveLeft(&id[threadIdx.x]);
		moveUp(&id[threadIdx.x]);
		break;
	case 1:
		moveUp(&id[threadIdx.x]);
		break;
	case 2:
		moveRight(&id[threadIdx.x]);
		moveUp(&id[threadIdx.x]);
		break;
	case 3:
		moveLeft(&id[threadIdx.x]);
		break;
	case 4:
		moveRight(&id[threadIdx.x]);
		break;
	case 5:
		moveLeft(&id[threadIdx.x]);
		moveDown(&id[threadIdx.x]);
		break;
	case 6:
		moveDown(&id[threadIdx.x]);
		break;
	case 7:
		moveRight(&id[threadIdx.x]);
		moveDown(&id[threadIdx.x]);
		break;
	}
	if (id[threadIdx.x] >= sizeOfArray || id[threadIdx.x] < 0)
		return;

	surroundState[threadIdx.x] = boxesStatusIn[id[threadIdx.x]];

	__syncthreads();

	if (threadIdx.x == 0)
	{
		for (bool state : surroundState)
		{
			sum += state;
		}
		if (state)
		{
			if (lifeArray[sum])
				boxesStatusOut[boxId] = true;
			else
				boxesStatusOut[boxId] = false;
		}
		else
		{
			if (deathArray[sum])
				boxesStatusOut[boxId] = true;
			else
				boxesStatusOut[boxId] = false;
		}
	}
}

void gpu::flipCellStatus(unsigned long long int x, GpuData& gpu)
{
	hipMemcpy(gpu.cellsStatusBuffer_p, gpu.dev_cellsStatusIn_p, gpu.cellsStatusLength * sizeof(bool), hipMemcpyDeviceToHost);
	gpu.cellsStatusBuffer_p[x] = !gpu.cellsStatusBuffer_p[x];
	hipMemcpy(gpu.dev_cellsStatusIn_p, gpu.cellsStatusBuffer_p, gpu.cellsStatusLength * sizeof(bool), hipMemcpyHostToDevice);
}

void gpu::setGameRules(GpuData& gpu, bool* lifeArray, bool* deathArray)
{
	hipMemcpy(gpu.lifeArray, lifeArray, 9 * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(gpu.deathArray, deathArray, 9 * sizeof(bool), hipMemcpyHostToDevice);
}

void gpu::freeMemory(GpuData& gpu)
{
	fprintf(stderr, "cuda free memory!");
	hipFree(gpu.dev_cellsStatusOut_p);
	hipFree(gpu.dev_cellsStatusIn_p);
	hipFree(gpu.lifeArray);
	hipFree(gpu.deathArray);
	// nie działa idk czemu
	if (gpu.cellsStatusBuffer_p) {
		free(gpu.cellsStatusBuffer_p);
		gpu.cellsStatusBuffer_p = 0;
	}
	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
}

hipError_t gpu::mallocMemory(GpuData& gpu)
{
	gpu.cellsStatusBuffer_p = (bool*)malloc(gpu.cellsStatusLength * sizeof(bool));
	memset(gpu.cellsStatusBuffer_p, 0, gpu.cellsStatusLength);

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&gpu.dev_cellsStatusIn_p, gpu.cellsStatusLength * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&gpu.lifeArray, 9 * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	bool* statusArray = (bool*)calloc(gpu.cellsStatusLength, sizeof(bool));
	cudaStatus = hipMemcpy(gpu.lifeArray, statusArray, 9 * sizeof(bool), hipMemcpyHostToDevice);
	free(statusArray);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&gpu.deathArray, 9 * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory(gpu);
		return cudaStatus;
	}


	cudaStatus = hipMemcpy(gpu.deathArray, statusArray, 9 * sizeof(bool), hipMemcpyHostToDevice);
	free(statusArray);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}



	cudaStatus = hipMalloc((void**)&gpu.dev_cellsStatusOut_p, gpu.cellsStatusLength * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	statusArray = (bool*)calloc(gpu.cellsStatusLength, sizeof(bool));
	cudaStatus = hipMemcpy(gpu.dev_cellsStatusIn_p, statusArray, gpu.cellsStatusLength * sizeof(bool), hipMemcpyHostToDevice);
	free(statusArray);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	return hipSuccess;
}

hipError_t gpu::calculateWithCuda(bool* statusArray, GpuData& gpu)
{
	hipError_t cudaStatus;
	/* for debug
	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	std::cout << "using " << properties.multiProcessorCount << " multiprocessors" << std::endl;
	std::cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << std::endl;*/
	std::cout << gpu.dev_cellsStatusIn_p << ", " << gpu.dev_cellsStatusOut_p << ", " << gpu.cellsStatusRowLength << ", " << gpu.cellsStatusLength << ", " << gpu.lifeArray << ", " << gpu.deathArray << std::endl;//*/
	calculateKernel << <gpu.cellsStatusLength, 8 >> > (gpu.dev_cellsStatusIn_p, gpu.dev_cellsStatusOut_p, gpu.cellsStatusRowLength, gpu.cellsStatusLength, gpu.lifeArray, gpu.deathArray);


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(statusArray, gpu.dev_cellsStatusOut_p, gpu.cellsStatusLength * sizeof(bool), hipMemcpyDeviceToHost);

	bool* buf = gpu.dev_cellsStatusIn_p;
	gpu.dev_cellsStatusIn_p = gpu.dev_cellsStatusOut_p;
	gpu.dev_cellsStatusOut_p = buf;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	return cudaStatus;
}

hipError_t gpu::getCellArray(bool* statusArray, GpuData& gpu) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(statusArray, gpu.dev_cellsStatusIn_p, gpu.cellsStatusLength * sizeof(bool), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	return cudaStatus;
}
