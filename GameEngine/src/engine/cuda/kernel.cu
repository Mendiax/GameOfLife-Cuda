#include "hip/hip_runtime.h"
﻿#include <engine/cuda/kernel.cuh>
#include <engine/board.h>
#include ""
#include <stdio.h>
#include <malloc.h>
#include <iostream>


__device__ __host__ void gpu::moveUp(unsigned long long int* id, unsigned long long int rowCount, unsigned long long int sizeOfBoxArray)
{
	if (*id + rowCount < sizeOfBoxArray)
		*id += rowCount;
	else
		*id %= rowCount;
}
__device__ __host__ void gpu::moveDown(unsigned long long int* id, unsigned long long int rowCount, unsigned long long int sizeOfBoxArray)
{
	if (*id >= rowCount)
		*id -= rowCount;
	else
		*id = (*id + sizeOfBoxArray) - rowCount;
}
__device__ __host__ void gpu::moveLeft(unsigned long long int* id, unsigned long long int rowCount, unsigned long long int sizeOfBoxArray)
{
	if (0 == *id % rowCount)
		*id = *id + (rowCount - 1);
	else
		*id -= 1;
}
__device__ __host__ void gpu::moveRight(unsigned long long int* id, unsigned long long int rowCount, unsigned long long int sizeOfBoxArray)
{
	if (rowCount == *id % rowCount + 1)
		*id = *id + 1 - rowCount;
	else
		*id += 1;
}

__global__ void gpu::calculateKernel(bool* boxesStatusIn, bool* boxesStatusOut, unsigned long long int row, unsigned long long int sizeOfArray, bool* lifeArray, bool* deathArray)
{
	__shared__ long long int boxId;
	__shared__ unsigned int sum;
	__shared__ bool state;
	__shared__ unsigned long long int id[8];

	if (threadIdx.x == 0)
	{
		boxId = blockIdx.x;
		state = boxesStatusIn[boxId];
		sum = 0;
	}

	__syncthreads();

	id[threadIdx.x] = boxId;



	__shared__  bool surroundState[8];
	switch (threadIdx.x)
	{
	case 0:
		moveLeft(&id[threadIdx.x],row, sizeOfArray);
		moveUp(&id[threadIdx.x], row, sizeOfArray);
		break;
	case 1:
		moveUp(&id[threadIdx.x], row, sizeOfArray);
		break;
	case 2:
		moveRight(&id[threadIdx.x], row, sizeOfArray);
		moveUp(&id[threadIdx.x], row, sizeOfArray);
		break;
	case 3:
		moveLeft(&id[threadIdx.x], row, sizeOfArray);
		break;
	case 4:
		moveRight(&id[threadIdx.x], row, sizeOfArray);
		break;
	case 5:
		moveLeft(&id[threadIdx.x], row, sizeOfArray);
		moveDown(&id[threadIdx.x], row, sizeOfArray);
		break;
	case 6:
		moveDown(&id[threadIdx.x], row, sizeOfArray);
		break;
	case 7:
		moveRight(&id[threadIdx.x], row, sizeOfArray);
		moveDown(&id[threadIdx.x], row, sizeOfArray);
		break;
	}
	if (id[threadIdx.x] >= sizeOfArray || id[threadIdx.x] < 0)
		return;

	surroundState[threadIdx.x] = boxesStatusIn[id[threadIdx.x]];

	__syncthreads();

	if (threadIdx.x == 0)
	{
		for (bool state : surroundState)
		{
			sum += state;
		}
		if (state)
		{
			if (lifeArray[sum])
				boxesStatusOut[boxId] = true;
			else
				boxesStatusOut[boxId] = false;
		}
		else
		{
			if (deathArray[sum])
				boxesStatusOut[boxId] = true;
			else
				boxesStatusOut[boxId] = false;
		}
	}
}

void gpu::flipCellStatus(unsigned long long int x, GpuData& gpu)
{
	hipMemcpy(gpu.cellsStatusBuffer_p, gpu.dev_cellsStatusIn_p, gpu.cellsStatusLength * sizeof(bool), hipMemcpyDeviceToHost);
	gpu.cellsStatusBuffer_p[x] = !gpu.cellsStatusBuffer_p[x];
	hipMemcpy(gpu.dev_cellsStatusIn_p, gpu.cellsStatusBuffer_p, gpu.cellsStatusLength * sizeof(bool), hipMemcpyHostToDevice);
}

void gpu::setGameRules(GpuData& gpu, bool* lifeArray, bool* deathArray)
{
	hipMemcpy(gpu.lifeArray, lifeArray, 9 * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(gpu.deathArray, deathArray, 9 * sizeof(bool), hipMemcpyHostToDevice);
}

void gpu::freeMemory(GpuData& gpu)
{
	fprintf(stderr, "cuda free memory!");
	hipFree(gpu.dev_cellsStatusOut_p);
	hipFree(gpu.dev_cellsStatusIn_p);
	hipFree(gpu.lifeArray);
	hipFree(gpu.deathArray);
	// nie działa idk czemu
	if (gpu.cellsStatusBuffer_p) {
		free(gpu.cellsStatusBuffer_p);
		gpu.cellsStatusBuffer_p = 0;
	}
	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
}

hipError_t gpu::mallocMemory(GpuData& gpu)
{
	gpu.cellsStatusBuffer_p = (bool*)malloc(gpu.cellsStatusLength * sizeof(bool));
	memset(gpu.cellsStatusBuffer_p, 0, gpu.cellsStatusLength);

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&gpu.dev_cellsStatusIn_p, gpu.cellsStatusLength * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&gpu.lifeArray, 9 * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	bool* statusArray = (bool*)calloc(gpu.cellsStatusLength, sizeof(bool));
	cudaStatus = hipMemcpy(gpu.lifeArray, statusArray, 9 * sizeof(bool), hipMemcpyHostToDevice);
	free(statusArray);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&gpu.deathArray, 9 * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory(gpu);
		return cudaStatus;
	}


	cudaStatus = hipMemcpy(gpu.deathArray, statusArray, 9 * sizeof(bool), hipMemcpyHostToDevice);
	free(statusArray);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}



	cudaStatus = hipMalloc((void**)&gpu.dev_cellsStatusOut_p, gpu.cellsStatusLength * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	statusArray = (bool*)calloc(gpu.cellsStatusLength, sizeof(bool));
	cudaStatus = hipMemcpy(gpu.dev_cellsStatusIn_p, statusArray, gpu.cellsStatusLength * sizeof(bool), hipMemcpyHostToDevice);
	free(statusArray);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	return hipSuccess;
}

hipError_t gpu::calculateWithCuda(bool* statusArray, GpuData& gpu)
{
	hipError_t cudaStatus;
	/* for debug
	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	std::cout << "using " << properties.multiProcessorCount << " multiprocessors" << std::endl;
	std::cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << std::endl;*/
	std::cout << gpu.dev_cellsStatusIn_p << ", " << gpu.dev_cellsStatusOut_p << ", " << gpu.cellsStatusRowLength << ", " << gpu.cellsStatusLength << ", " << gpu.lifeArray << ", " << gpu.deathArray << std::endl;//*/
	calculateKernel << <gpu.cellsStatusLength, 8 >> > (gpu.dev_cellsStatusIn_p, gpu.dev_cellsStatusOut_p, gpu.cellsStatusRowLength, gpu.cellsStatusLength, gpu.lifeArray, gpu.deathArray);


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freeMemory(gpu);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(statusArray, gpu.dev_cellsStatusOut_p, gpu.cellsStatusLength * sizeof(bool), hipMemcpyDeviceToHost);

	bool* buf = gpu.dev_cellsStatusIn_p;
	gpu.dev_cellsStatusIn_p = gpu.dev_cellsStatusOut_p;
	gpu.dev_cellsStatusOut_p = buf;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	return cudaStatus;
}

hipError_t gpu::getCellArray(bool* statusArray, GpuData& gpu) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(statusArray, gpu.dev_cellsStatusIn_p, gpu.cellsStatusLength * sizeof(bool), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeMemory(gpu);
		return cudaStatus;
	}

	return cudaStatus;
}
